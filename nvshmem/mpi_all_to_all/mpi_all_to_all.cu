#include "hip/hip_runtime.h"
#include "../../cuda_util/cuda_util.h"
#include "../../cuda_util/random_fill.h"
#include "../../util/argparse.h"
#include "../../util/mpi_util.h"
#include "../../util/simple_utils.h"
#include "host/nvshmem_coll_api.h"
#include "mpi.h"
#include "nvshmem.h"
#include "nvshmemx.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <time.h>
#include <unistd.h>

#define CUDA_CHECK(stmt)                                                       \
  do {                                                                         \
    hipError_t result = (stmt);                                               \
    if (hipSuccess != result) {                                               \
      fprintf(stderr, "[%s:%d] CUDA failed with %s \n", __FILE__, __LINE__,    \
              hipGetErrorString(result));                                     \
      exit(-1);                                                                \
    }                                                                          \
  } while (0)

static struct options opts;
static struct parser_doc parser_doc;

clock_t start, endparse, cusetup, endwarmup, enditer, c_end;

void bench_iter(int nDev, void *sendbuff, void **recvbuff, int size,
                int data_type, hipStream_t s);

__global__ void all_to_all_kernel(void *sendbuff, void **recvbuff, int size,
                                  int data_type) {
  int mype = nvshmem_my_pe();
  int npes = nvshmem_n_pes();

  for (int peer = 0; peer < npes; peer++) {
    if (peer != mype) {
      switch (data_type) {
      case options::OPTION_CHAR:
        nvshmem_char_put((char *)recvbuff[mype], (const char *)sendbuff, size,
                         peer);
        break;
      case options::OPTION_INT:
        nvshmem_int_put((int *)recvbuff[mype], (const int *)sendbuff, size,
                        peer);
        break;
      case options::OPTION_FLOAT:
        nvshmem_float_put((float *)recvbuff[mype], (const float *)sendbuff,
                          size, peer);
        break;
      }
    }
  }
}

int main(int argc, char *argv[]) {
  start = clock();
  build_parser_doc("MPI all to all with nvshmem", "", "1",
                   "egencer20@ku.edu.tr", &parser_doc);
  argument_parse(&opts, &parser_doc, argc, argv);

  int myRank, nRanks, localRank = 0;
  int size = opts.data_len;

  int data_size = 0;
  int data_type = opts.data_type;

  switch (opts.data_type) {
  case options::OPTION_CHAR:
    data_size = sizeof(char);
    break;
  case options::OPTION_FLOAT:
    data_size = sizeof(float);
    break;
  case options::OPTION_INT:
    data_size = sizeof(int);
    break;
  }

  int mype_node;
  hipStream_t stream;
  MPI_Comm mpi_comm = MPI_COMM_WORLD;
  nvshmemx_init_attr_t attr;

  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MPI_COMM_WORLD, &myRank);
  MPI_Comm_size(MPI_COMM_WORLD, &nRanks);

  attr.mpi_comm = &mpi_comm;
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
  mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
  int nDev = nRanks;

  void *sendbuff;
  void *recvbuff[nRanks];

  REPORT("NDEV: %d myrank: %d\n", nDev, mype_node);
  report_options(&opts);
  endparse = clock();

  CUDA_CHECK(hipSetDevice(mype_node));
  CUDA_CHECK(hipStreamCreate(&stream));

  void *send_buffer = nvshmem_malloc(data_size * size);

  for (int i = 0; i < nDev; i++)
    recvbuff[i] = nvshmem_malloc(data_size * size);

  // void *tmp = malloc(data_size * size);
  // memset(tmp, 0, data_size * size);
  random_fill_host(send_buffer, data_size * size);

  // CUDA_CHECK(hipMemcpyAsync(send_buffer, tmp, data_size * size,
  //                            hipMemcpyHostToDevice, stream));
  nvshmem_barrier_all();
  CUDA_CHECK(hipStreamSynchronize(stream));

  free(tmp);

  cusetup = clock();

  for (int iter = 0; iter < opts.warmup_iterations; iter++) {
    bench_iter(nDev, sendbuff, recvbuff, size, data_type, stream);
  }

  endwarmup = clock();

  for (int iter = 0; iter < opts.iterations; iter++) {
    bench_iter(nDev, sendbuff, recvbuff, size, data_type, stream);
  }

  enditer = clock();

  // free device buffers

  nvshmem_free(sendbuff);
  for (int i = 0; i < nDev; i++)
    nvshmem_free(recvbuff[i]);

  MPICHECK(MPI_Finalize());

  c_end = clock();

#define CLOCK_CONVERT(x) (((double)x) / CLOCKS_PER_SEC)

  REPORT("Completed Succesfully\n"
         "parsing arguments: %.2f\n"
         "cuda setup: %.2f\n"
         "warmup, avg: %.2f, %.2f\n"
         "iterations, avg: %.2f, %.2f\n"
         "cleanup: %.2f\n"
         "total: %.2f\n\n",
         CLOCK_CONVERT(endparse - start), CLOCK_CONVERT(cusetup - endparse),
         CLOCK_CONVERT(endwarmup - cusetup),
         (CLOCK_CONVERT(endwarmup - cusetup)) /
             (opts.warmup_iterations > 0 ? opts.warmup_iterations : 1),
         CLOCK_CONVERT(enditer - endwarmup),
         (CLOCK_CONVERT(enditer - endwarmup)) /
             (opts.iterations > 0 ? opts.iterations : 1),
         CLOCK_CONVERT(c_end - enditer), CLOCK_CONVERT(c_end - start));
  return 0;
}

void bench_iter(int nDev, void *sendbuff, void **recvbuff, int size,
                int data_type, hipStream_t stream) {

  // start the kernel in each iteration
  all_to_all_kernel<<<1, 1, 0, stream>>>(sendbuff, recvbuff, size, data_type);
  nvshmemx_barrier_all_on_stream(stream);
}